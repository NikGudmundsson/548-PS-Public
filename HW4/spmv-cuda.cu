#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "cmdline.h"
#include "input.h"
#include "config.h"
#include "timer.h"
#include "formats.h"

#define max(a,b) \
({ __typeof__ (a) _a = (a); \
   __typeof__ (b) _b = (b); \
 _a > _b ? _a : _b; })

#define min(a,b) \
({ __typeof__ (a) _a = (a); \
   __typeof__ (b) _b = (b); \
 _a < _b ? _a : _b; })

void usage(int argc, char** argv)
{
    printf("Usage: %s [my_matrix.mtx]\n", argv[0]);
    printf("Note: my_matrix.mtx must be real-valued sparse matrix in the MatrixMarket file format.\n"); 
}

__global__ void benchmark_coo_spmv(int nonzero, int* rows, int* cols, float* vals, float* x, float* y) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    //int num_nonzeros = nonzero;

    if (idx < nonzero) {
        // warmup    
        y[rows[idx]] += vals[idx] * x[cols[idx]];

        // 500 Iterations
        int num_iterations;
        num_iterations = 500;

        //printf("\tPerforming %d iterations\n", num_iterations);
        
        for(int j = 0; j < num_iterations; j++) {
            y[rows[idx]] += vals[idx] * x[cols[idx]];
        }

        //printf("I am %d!\n", idx);
    }
}

int main(int argc, char** argv)
{
    if (get_arg(argc, argv, "help") != NULL){
        usage(argc, argv);
        return 0;
    }

    char * mm_filename = NULL;
    if (argc == 1) {
        printf("Give a MatrixMarket file.\n");
        return -1;
    } else 
        mm_filename = argv[1];

    coo_matrix coo;
    read_coo_matrix(&coo, mm_filename);

    // fill matrix with random values: some matrices have extreme values, 
    // which makes correctness testing difficult, especially in single precision
    srand(13);
    for(int i = 0; i < coo.num_nonzeros; i++) {
        coo.vals[i] = 1.0 - 2.0 * (rand() / (RAND_MAX + 1.0)); 
        // coo.vals[i] = 1.0;
    }
    
    printf("\nfile=%s rows=%d cols=%d nonzeros=%d\n", mm_filename, coo.num_rows, coo.num_cols, coo.num_nonzeros);
    fflush(stdout);

#ifdef TESTING
//print in COO format
    printf("Writing matrix in COO format to test_COO ...");
    FILE *fp = fopen("test_COOcu", "w");
    fprintf(fp, "%d\t%d\t%d\n", coo.num_rows, coo.num_cols, coo.num_nonzeros);
    fprintf(fp, "coo.rows:\n");
    for (int i=0; i<coo.num_nonzeros; i++)
    {
      fprintf(fp, "%d  ", coo.rows[i]);
    }
    fprintf(fp, "\n\n");
    fprintf(fp, "coo.cols:\n");
    for (int i=0; i<coo.num_nonzeros; i++)
    {
      fprintf(fp, "%d  ", coo.cols[i]);
    }
    fprintf(fp, "\n\n");
    fprintf(fp, "coo.vals:\n");
    for (int i=0; i<coo.num_nonzeros; i++)
    {
      fprintf(fp, "%f  ", coo.vals[i]);
    }
    fprintf(fp, "\n");
    fclose(fp);
    printf("... done!\n");
#endif 

    //initialize host arrays
    float * x = (float*)malloc(coo.num_cols * sizeof(float));
    float * y = (float*)malloc(coo.num_rows * sizeof(float));

    for(int i = 0; i < coo.num_cols; i++) {
        x[i] = rand() / (RAND_MAX + 1.0); 
        // x[i] = 1;
    }

    //printf("38th value x %f\n", x[37]);

    for(int i = 0; i < coo.num_rows; i++)
        y[i] = 0;

    // printf("38th value y %f\n", y[37]);
    // printf("Bytes %d vs bytes 2 %d vs bytes 3 %d\n", coo.num_cols*sizeof(float), sizeof(x), sizeof(y));

    float* yGPU = NULL;
    float* xGPU = NULL;
    int* colsGPU = NULL;
    int* rowsGPU = NULL;
    float* valsGPU = NULL;

    // Allocate space on GPU for copies of the data
    if (hipMalloc((void**)&yGPU, coo.num_rows*sizeof(float)) != hipSuccess) {
        printf("fail 1\n");
        exit(1);
    }

    if (hipMalloc((void**)&xGPU, coo.num_cols*sizeof(float)) != hipSuccess) {
        printf("fail 2\n");
        exit(1);
    }
    
    if (hipMalloc((void**)&colsGPU, coo.num_nonzeros*sizeof(int)) != hipSuccess) {
        printf("fail 3\n");
        exit(1);
    }

    if (hipMalloc((void**)&rowsGPU, coo.num_nonzeros*sizeof(int)) != hipSuccess) {
        printf("fail 4\n");
        exit(1);
    }

    if (hipMalloc((void**)&valsGPU, coo.num_nonzeros*sizeof(float)) != hipSuccess) {
        printf("fail 5\n");
        exit(1);
    }

    // Copy Sequences over to the device
    if (hipMemcpy(yGPU, y, coo.num_rows*sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
        printf("fail 6\n");
        exit(1);
    }

    if (hipMemcpy(xGPU, x, coo.num_cols*sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
        printf("fail 7\n");
        exit(1);
    }

    if (hipMemcpy(colsGPU, coo.cols, coo.num_nonzeros*sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
        printf("fail 8\n");
        exit(1);
    }

    if (hipMemcpy(rowsGPU, coo.rows, coo.num_nonzeros*sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
        printf("fail 9\n");
        exit(1);
    }

    if (hipMemcpy(valsGPU, coo.vals, coo.num_nonzeros*sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
        printf("fail 10\n");
        exit(1);
    }

    /* Benchmarking */
    // Block and grid dimensions.
    int threadsPerBlock = 100;
    // Round up for the number of blocks we need.
    int blocksPerGrid = ( coo.num_nonzeros + threadsPerBlock - 1 ) / threadsPerBlock;

    // 200 Iterations
    int num_iterations;
    num_iterations = 500;

    timer t;
    timer_start(&t);

    benchmark_coo_spmv<<<blocksPerGrid, threadsPerBlock>>>(coo.num_nonzeros, rowsGPU, colsGPU, valsGPU, xGPU, yGPU);
    if (hipGetLastError() != hipSuccess) {
        printf("fail 11\n");
        exit(1);
    }
    
    double msec_per_iteration = milliseconds_elapsed(&t) / (double) num_iterations;
    //double sec_per_iteration = msec_per_iteration / 1000.0;
    //double GFLOPs = (sec_per_iteration == 0) ? 0 : (2.0 * (double) coo->num_nonzeros / sec_per_iteration) / 1e9;
    //double GBYTEs = (sec_per_iteration == 0) ? 0 : ((double) bytes_per_coo_spmv(coo) / sec_per_iteration) / 1e9;
    printf("\tbenchmarking COO-SpMV: %8.10f ms\n", msec_per_iteration); 

    // Copy Back Results

    float * hostx = (float*)malloc(coo.num_cols * sizeof(float));
    float * hosty = (float*)malloc(coo.num_rows * sizeof(float));
    // printf("Col num %d\n", coo.num_cols);
    // printf("Bytes %d vs bytes 2 %d vs bytes 3 %d\n", coo.num_cols*sizeof(float), sizeof(x[0])*coo.num_cols, sizeof(hostx[0])*coo.num_cols);
    int err = hipMemcpy(hostx, xGPU, coo.num_cols*sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("%d fail 12\n", err);
        exit(1);
    }

    if (hipMemcpy(hosty, yGPU, coo.num_rows*sizeof(float), hipMemcpyDeviceToHost) != hipSuccess) {
        printf("fail 13\n");
        exit(1);
    }

    // Free CUDA Memory
    hipFree(xGPU);
    hipFree(yGPU);
    hipFree(colsGPU);
    hipFree(rowsGPU);
    hipFree(valsGPU);

    hipDeviceReset();

    // double coo_gflops;
    // coo_gflops = benchmark_coo_spmv(&coo, x, y);

    /* Test correctnesss */
#ifdef TESTING
    printf("Writing x and y vectors ...");
    fp = fopen("test_xcu2", "w");
    for (int i=0; i<coo.num_cols; i++)
    {
      fprintf(fp, "%f\n", hostx[i]);
    }
    fclose(fp);
    fp = fopen("test_ycu2", "w");
    for (int i=0; i<coo.num_rows; i++)
    {
      fprintf(fp, "%f\n", hosty[i]);
    }
    fclose(fp);
    printf("... done!\n");
#endif

    delete_coo_matrix(&coo);
    free(hostx);
    free(hosty);
    free(x);
    free(y);

    return 0;
}

